#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <fstream>
#include <string>
#include <cmath>
#include <climits>
#include <iostream>
#include <chrono>


using namespace std;

#define MAX_THREAD_PER_BLOCK 512
#define DEBUG_PRINT

/*
  PARALLEL NN - VERSION 2
*/

typedef unsigned short int usint;

const usint num_dimensions = 16;
const usint numPointsTest = 1000;
const usint numPointsTrain = 19000;
const usint streamCount = 4;

struct Coordinates {
	usint points[num_dimensions];
};

__device__ float getDistance(const Coordinates & coord1, const Coordinates & coord2) {
	float square_sum = 0;
	for (int i = 0; i < num_dimensions; i++) {
		const int c1 = coord1.points[i];
		const int c2 = coord2.points[i];
		square_sum += (c1 - c2)*(c1 - c2);
	}
	return sqrt(square_sum);
}

__global__ void nearestNeighbor(Coordinates * trainCoords, Coordinates * testCoords, const usint sizeTest, const usint sizeTrain, usint * nearestNeighbors) {
	const usint threadId = blockIdx.x*blockDim.x + threadIdx.x;
	if (threadId < sizeTest) { // DEBUG
		usint nearestNeighbor = 0;
		usint nearestDistance = USHRT_MAX;
		for (int trainCoordInd = 0; trainCoordInd < sizeTrain; trainCoordInd++) {
			float currentDistance = getDistance(trainCoords[trainCoordInd], testCoords[threadId]);
			if (currentDistance < nearestDistance) {
				nearestNeighbor = trainCoordInd;
				nearestDistance = currentDistance;
			}
		}
		nearestNeighbors[threadId] = nearestNeighbor;
	}
}

bool checkError(const hipError_t & error, const char * msg = "") {
	if (error != hipSuccess) {
		printf("CUDA ERROR: %s\n", msg);
		cout << error << endl;
		exit(1);
	}
	return true;
}

int main() {
	// 1 - INITIALIZE READ STREAMS
	const char * testFile = "test.txt";
	const char * trainFile = "train.txt";
	FILE * test_is = fopen(testFile, "r"), * train_is = fopen(trainFile, "r");
	if (!test_is) {
		cerr << "Cannot open " << testFile << endl;
		exit(1);
	}
	if (!train_is) {
		cerr << "Cannot open " << trainFile << endl;
		exit(1);
	}

	hipSetDevice(0); // initialize CUDA context
	cout << "\t--------------------\n";
	chrono::high_resolution_clock::time_point begin = chrono::high_resolution_clock::now(), temp, end;

	// 2 - SET EXECUTION PARAMETERS
	hipStream_t streams[streamCount]; // create four CUDA streams
	hipError_t hipError_t;

	usint numThreadsPerBlock = numPointsTest;
	usint numBlocks = 1;
	if (numPointsTest > MAX_THREAD_PER_BLOCK) {
		numBlocks = std::ceil(static_cast<double>(numPointsTest) / MAX_THREAD_PER_BLOCK);
		numThreadsPerBlock = MAX_THREAD_PER_BLOCK;
	}
	numThreadsPerBlock /= streamCount;
	cout << "Kernels will be called with " << numBlocks << " blocks with " << numThreadsPerBlock << " threads each\n";

	// 3 - READ TRAIN COORDINATES FROM FILE STREAMS
	// device pointers
	Coordinates * d_testCoordinates[streamCount], *d_trainCoordinates;
	usint * d_nearestNeighbors[streamCount];
	// host pointers
	Coordinates * h_testCoordinates[streamCount], *h_trainCoordinates;
	usint * h_nearestNeighbors[streamCount];

	hipError_t = hipHostMalloc((void**)&h_trainCoordinates, numPointsTrain * sizeof(Coordinates), hipHostMallocDefault);
	checkError(hipError_t, "cudamallochost - h_trainCoordinates");

	// read train points to host
	for (int i = 0; i < numPointsTrain; i++) {
		fscanf(train_is, "%d, %d, %d, %d, %d, %d, %d, %d, %d, %d, %d, %d, %d, %d, %d, %d\n", &h_trainCoordinates[i].points[0], &h_trainCoordinates[i].points[1], &h_trainCoordinates[i].points[2],
			&h_trainCoordinates[i].points[3], &h_trainCoordinates[i].points[4], &h_trainCoordinates[i].points[5], &h_trainCoordinates[i].points[6], &h_trainCoordinates[i].points[7],
			&h_trainCoordinates[i].points[8], &h_trainCoordinates[i].points[9], &h_trainCoordinates[i].points[10], &h_trainCoordinates[i].points[11], &h_trainCoordinates[i].points[12],
			&h_trainCoordinates[i].points[13], &h_trainCoordinates[i].points[14], &h_trainCoordinates[i].points[15]);
	}
	cout << "done reading training coordinates to host pinned memory" << endl;

	// copy train coordinates to device
	hipError_t = hipMalloc((void**)&d_trainCoordinates, numPointsTrain * sizeof(Coordinates));
	checkError(hipError_t, "hipMalloc - d_trainCoordinates");
	hipError_t = hipMemcpy(d_trainCoordinates, h_trainCoordinates, numPointsTrain * sizeof(Coordinates), hipMemcpyHostToDevice);
	checkError(hipError_t, "hipMemcpyAsync - d_trainCoordinates");
	
	chrono::high_resolution_clock::time_point kernel_start = chrono::high_resolution_clock::now();
	for (usint stream = 0; stream < streamCount; stream++) {
		// 1 - create stream
		hipStreamCreate(&streams[stream]);

		// 2 - Host memory - allocate memory on host for results and test coordinates
		hipError_t = hipHostMalloc((void**)&h_nearestNeighbors[stream], (numPointsTest / streamCount) * sizeof(usint), hipHostMallocDefault);
		checkError(hipError_t, "cudamallochost - h_nearestneighbors");
		hipError_t = hipHostMalloc((void**)&h_testCoordinates[stream], (numPointsTest / streamCount) * sizeof(Coordinates), hipHostMallocDefault);
		checkError(hipError_t, "cudamallochost - h_testCoordinates");

		// 3 - Host memory - read test points
		for (int i = 0; i < numPointsTest / streamCount; i++) {
			fscanf(test_is, "%d, %d, %d, %d, %d, %d, %d, %d, %d, %d, %d, %d, %d, %d, %d, %d", &h_testCoordinates[stream][i].points[0], &h_testCoordinates[stream][i].points[1], &h_testCoordinates[stream][i].points[2],
				&h_testCoordinates[stream][i].points[3], &h_testCoordinates[stream][i].points[4], &h_testCoordinates[stream][i].points[5], &h_testCoordinates[stream][i].points[6], &h_testCoordinates[stream][i].points[7],
				&h_testCoordinates[stream][i].points[8], &h_testCoordinates[stream][i].points[9], &h_testCoordinates[stream][i].points[10], &h_testCoordinates[stream][i].points[11], &h_testCoordinates[stream][i].points[12],
				&h_testCoordinates[stream][i].points[13], &h_testCoordinates[stream][i].points[14], &h_testCoordinates[stream][i].points[15]);
		}
		
		// 4 - Device memory - allocate space for test coordiantes and result array for this stream to write its results to
		hipError_t = hipMalloc((void**)&d_testCoordinates[stream], (numPointsTest / streamCount) * sizeof(Coordinates));
		checkError(hipError_t, "hipMalloc - d_testCoordiantes");
		hipError_t = hipMalloc((void**)&d_nearestNeighbors[stream], (numPointsTest / streamCount) * sizeof(usint));
		checkError(hipError_t, "hipMalloc - d_nearestNeighbors");
	
		// 5 - copy test coordinates to device in async
		temp = chrono::high_resolution_clock::now();
		hipError_t = hipMemcpyAsync(d_testCoordinates[stream], h_testCoordinates[stream], (numPointsTest / streamCount) * sizeof(Coordinates), hipMemcpyHostToDevice, streams[stream]);
		checkError(hipError_t, "hipMemcpy - d_testCoordinates");
		end = chrono::high_resolution_clock::now();
		cout << "data copied to device memory [" << chrono::duration_cast<chrono::milliseconds>(end - temp).count() << " ms]\n"
			<< "executing kernel with " << numBlocks << " blocks with " << numThreadsPerBlock << " threads each" << endl;
		
		// 6 - Inovke kernel for current stream
		usint *& currentResultArray = d_nearestNeighbors[stream];
		nearestNeighbor<<< numBlocks, numThreadsPerBlock, 0, streams[stream] >>>(d_trainCoordinates, d_testCoordinates[stream], numPointsTest / streamCount, numPointsTrain, currentResultArray);
		hipError_t = hipMemcpyAsync(h_nearestNeighbors[stream], d_nearestNeighbors[stream], (numPointsTest / streamCount) * sizeof(usint), hipMemcpyDeviceToHost, streams[stream]);
		checkError(hipError_t, "hipMemcpy - h_nearestNeighbors");
	}
	
	// Wait for GPU to terminate and fetch results
	hipError_t = hipGetLastError();
	checkError(hipError_t, "before deviceSync() error!");
	hipDeviceSynchronize();
	end = chrono::high_resolution_clock::now();
	cout << "Computation + read test data: " << chrono::duration_cast<chrono::milliseconds>(end - kernel_start).count() << " ms\n";
	cout << "\t--------------------\n";
	end = chrono::high_resolution_clock::now();
	
	ofstream os("output.txt");
	
	for (int stream = 0; stream < streamCount; stream++) {
		for (int i = 0; i < numPointsTest / streamCount; i++) {
			os << h_nearestNeighbors[stream][i] << endl;
		}
	}
	
	end = chrono::high_resolution_clock::now();
	cout << "\t--------------------\nTotal time: " << chrono::duration_cast<chrono::milliseconds>(end - begin).count() << " ms\nterminating\n";
	
	return 0;
}
